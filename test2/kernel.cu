#include "hip/hip_runtime.h"
#include "mycommon.cuh"
#include "vec3.cuh"

template<typename T = float>
__device__ T operator/(T lhs, T rhs)
{
	return (rhs == 0) ? (T)MYINFINITY : lhs / rhs;
}

template<typename pT>
__host__ __device__ inline void swap(pT& a, pT& b)
{
	pT temp = a;
	a = b;
	b = temp;
}

template<typename T = float>
class samplingpos
{
public:
	T u = 0;
	T v = 0;
	__host__ __device__ samplingpos(T u = 0, T v = 0)
		:u(u), v(v)
	{}
	__host__ __device__ ~samplingpos()
	{}
};

//the grammar is: device can manipulate both ray objects and pointers
template<typename T = float>
class raysegment
{
public:
	vec3<T> pos, dir;
	samplingpos<T> spos;
	T intensity = 0; //radiant intensity in W/sr

	int status = 1; // 1 is active, 0 is deactive, 2 is finised, more to come

	__host__ __device__ raysegment(const vec3<T>& pos = vec3<T>(0, 0, 0), const vec3<T>& dir = vec3<T>(0,0,-1), const samplingpos<T>& spos = samplingpos<T>(0,0),T intensity =0):
		pos(pos), dir(dir),spos(spos),intensity(intensity)
	{
		LOG1("ray segment created")
	}

	__host__ __device__ ~raysegment()
	{
		LOG1("ray segment destructor called")
	}

	template<typename T>
	friend std::ostream& operator<<(std::ostream& os, const raysegment<T>& rs);
};

template<typename T>
std::ostream& operator<<(std::ostream& os, const raysegment<T>& rs)
{
	os << "ray at " << rs.pos << " pointing " << rs.dir;
	return os;
}

//the grammar is: host manipulate the bundle objects, device can only manipulate the bundle pointers...
template <typename T>
class raybundle
{
public:
	int size; //modifiable by member function
	T wavelength = 555; //wavelength in nm
	raysegment<T>*prays = nullptr;
	samplingpos<T>*samplinggrid = nullptr;
	raybundle<T>* d_sibling = nullptr; //sibling to this bundle on the device
									   //one bundle can only have one sibling at a time
									   //attempt to create a new sibling will delete the existing one

	//constructor and destructor, creation is limited to host only, to transfer ray bundle to device...
	//...must create the device sibling of the bundle
	__host__ raybundle(int size = bundlesize, T wavelength = 555)
		:size(size),wavelength(wavelength)
	{
		prays = new raysegment<T>[size];
		samplinggrid = new samplingpos<T>[size];
	}
	__host__  ~raybundle()
	{
		if (d_sibling != nullptr) freesibling();
		delete[] prays;
		delete[] samplinggrid;
	}
	
	//copy constructor
	__host__ raybundle(const raybundle <T>& origin)
		:size(origin.size), wavelength(origin.wavelength)
	{
		prays = new raysegment<T>[size];
		samplinggrid = new samplingpos<T>[size];
		memcpy(prays, origin.prays, size * sizeof(raysegment<T>));
		memcpy(samplinggrid, origin.samplinggrid, size * sizeof(samplingpos<T>));
	}

	//copy assignment operator
	__host__ raybundle<T>& operator=(const raybundle<T>& origin)
	{
		size = origin.size;
		wavelength = origin.wavelength;
		prays = new raysegment<T>[size];
		samplinggrid = new samplingpos<T>[size];
		memcpy(prays, origin.prays, size * sizeof(raysegment<T>));
		memcpy(samplinggrid, origin.samplinggrid, size * sizeof(samplingpos<T>));
		return *this;
	}

	//free the current sibling
	__host__ void freesibling()
	{
		hipFree(d_sibling->prays);
		hipFree(d_sibling->samplinggrid);
		hipFree(d_sibling);
		d_sibling = nullptr;
	}

	//create the sibling raybundle on GPU, copy this raybundle to it, and return a device pointer
	__host__ raybundle<T>* copytosibling()
	{
		//delete the current sibling
		if (d_sibling != nullptr)
		{
			freesibling();
		}

		//allocate memory on device
		CUDARUN(hipMalloc((void**)&d_sibling, sizeof(raybundle<T>)));
		CUDARUN(hipMalloc((void**)&(d_sibling->prays), size * sizeof(raysegment<T>)));
		CUDARUN(hipMalloc((void**)&(d_sibling->samplinggrid), size * sizeof(samplingpos<T>)));

		//copy data to device
		d_sibling->size = size;
		d_sibling->wavelength = wavelength;
		CUDARUN(hipMemcpy(d_sibling->prays, prays, size * sizeof(raysegment<T>), hipMemcpyHostToDevice));
		CUDARUN(hipMemcpy(d_sibling->samplinggrid, samplinggrid, size * sizeof(samplingpos<T>), hipMemcpyHostToDevice));

		if (hipGetLastError() != hipSuccess)
		{
			freesibling();
			return nullptr;
		}
		else
			return d_sibling;
	}

	//copy the sibling bundle from GPU to this ray bundle, return a this pointer
	__host__ raybundle<T>* copyfromsibling()
	{
		if (d_sibling != nullptr)
		{
			//copy new data in, assume size and wavelength is correctly mirrored between siblings
			CUDARUN(hipMemcpy(prays, d_sibling->prays, size * sizeof(raysegment<T>), hipMemcpyDeviceToHost));
			CUDARUN(hipMemcpy(samplinggrid, d_sibling->samplinggrid, size * sizeof(samplingpos<T>), hipMemcpyDeviceToHost));
		}
		return this;
	}

	//simplest initializer: generate 1D parallel ray fan along vertical direction
	__host__ raybundle<T>& init_1D_parallel(vec3<T> dir, T diam, T z_position)
	{
		float step = diam / size;
		float start = -(diam / 2) + (step / 2);
		for (int i = 0; i < size; i++)
		{
			prays[i] = raysegment<T>(vec3<T>(start + step * i, 0, z_position), dir);
			samplinggrid[i] = samplingpos<T>(i - size / 2, 0);
			printf("i = %d, (u,v) = (%f,%f), pos = (%f,%f,%f), dir = (%f,%f,%f) \n", i
				, samplinggrid[i].u, samplinggrid[i].v
				, prays[i].pos.x, prays[i].pos.y, prays[i].pos.z
				, prays[i].dir.x, prays[i].dir.y, prays[i].dir.z);
		}
		return *this;
	}

	//more sophisticated 2D equi-spherical-area initializer, note: x is horizontal, y is vertical, z is towards observer
	__host__ raybundle<T>& init_2D_dualpolar(vec3<T> originpos, T min_horz, T max_horz, T min_vert, T max_vert, T step)
	{
		//clamping the limits to pi/2
		min_horz = (min_horz < -MYPI / 2) ? -MYPI / 2 : min_horz;
		min_vert = (min_vert < -MYPI / 2) ? -MYPI / 2 : min_vert;
		max_horz = (max_horz > MYPI / 2) ? MYPI / 2 : max_horz;
		max_vert = (max_vert > MYPI / 2) ? MYPI / 2 : max_vert;

		//checking the max and min limits, they must be at least one step apart
		min_horz = (min_horz > max_horz - step) ? (max_horz - step) : min_horz;
		min_vert = (min_vert > max_vert - step) ? (max_vert - step) : min_vert;
		

		int temp_size = static_cast<int>((max_horz / step - min_horz / step + 1)* 
			(max_vert / step - min_vert / step + 1));

		//for safety, reclean the object before initialization
		if (d_sibling != nullptr) freesibling();
		delete[] prays;
		delete[] samplinggrid;
		size = 0;

		//assign temporary memory
		raysegment<T>* temp_prays = new raysegment<T>[temp_size];
		samplingpos<T>* temp_samplinggrid = new samplingpos<T>[temp_size];

		//declaration
		T angle_horz;
		T angle_vert;
		T semi_axis_horz;
		T semi_axis_vert;
		T x, y, z;

		for (int i = static_cast<int>(min_horz / step); i < (max_horz / step)+1; i++)
		{
			for (int j = static_cast<int>(min_vert / step); j < (max_vert / step)+1; j++)
			{
				//if the sampling point is within ellipse-bound and smaller than pi/2
				angle_horz = i * step;
				angle_vert = j * step;
				semi_axis_horz = (angle_horz < 0) ? min_horz : max_horz;
				semi_axis_vert = (angle_vert < 0) ? min_vert : max_vert;
				if (((angle_horz / semi_axis_horz)*(angle_horz / semi_axis_horz) +
					(angle_vert / semi_axis_vert)*(angle_vert / semi_axis_vert)
					<= 1) 
					&& (angle_horz < MYPI/2 && angle_vert < MYPI/2)
					&& (sin(angle_horz)*sin(angle_horz)+sin(angle_vert)*sin(angle_vert)<=1)
					)
				{
					//register
					temp_samplinggrid[size] = samplingpos<T>(i, j);
					/*
					z = -1 / sqrt(1 + tan(angle_horz)*tan(angle_horz) + tan(angle_vert)*tan(angle_vert));
					x = -z * tan(angle_horz);
					y = -z * tan(angle_vert);
					*/
					x = sin(angle_horz);
					y = sin(angle_vert);
					z = sqrt(1 - x * x - y * y);
					temp_prays[size] = raysegment<T>(originpos, vec3<T>(x, y, z));
					size += 1;
				}
			}
		}

		//copy temporary arrays to final arrays
		prays = new raysegment<T>[size];
		samplinggrid = new samplingpos<T>[size];
		memcpy(prays, temp_prays, size * sizeof(raysegment<T>));
		memcpy(samplinggrid, temp_samplinggrid, size * sizeof(samplingpos<T>));
		delete[] temp_prays;
		delete[] temp_samplinggrid;

		//debugging: printout trace
#ifdef _DEBUGMODE2
		if (samplinggrid != nullptr && prays != nullptr)
		{
			for (int i = 0; i < size; i++)
			{
				printf("i = %d\t (u,v) = (%f,%f)\t at (%f,%f,%f)\t pointing (%f,%f,%f)\n", i,
					samplinggrid[i].u, samplinggrid[i].v,
					prays[i].pos.x, prays[i].pos.y, prays[i].pos.z,
					prays[i].dir.x, prays[i].dir.y, prays[i].dir.z);
			}
		}
		else printf("error: null ptr detected");
#endif

		return *this;
	}
};

template<typename T = float>
class mysurface
{
public:
	vec3<T> pos; // at first no rotation of axis
	T diameter; // default to 10 mm, see constructor
	int type; //0 is image surface, 1 is power surface, 2 is stop surface

	__host__ __device__ mysurface(const vec3<T>& pos = vec3<T>(0,0,0), T diameter = 10, int type = 0) :
		pos(pos), diameter(diameter), type(type)
	{
		LOG1("my surface created")
	}

	__host__ __device__ ~mysurface()
	{
		LOG1("surface destructor called")
	}

	//TO DO: needed a more sophisticated implementation of this hit box function
	// return true if position is inside hit box
	__host__ __device__ inline virtual bool hitbox(const vec3<T>& pos)
	{
		return ((pos.x*pos.x + pos.y*pos.y) <= (diameter*diameter / 4)) ? true : false;
	}

	__host__ __device__ inline virtual raysegment<T> coordinate_transform(const raysegment<T>& original)
	{
		return raysegment<T>(original.pos - this->pos, original.dir);
	}

	__host__ __device__ inline virtual raysegment<T> coordinate_detransform(const raysegment<T>& original)
	{
		return raysegment<T>(original.pos + this->pos, original.dir);
	}

	__host__ __device__ virtual int size()
	{
		return sizeof(*this);
	}
};

template<typename T = float>
class powersurface:public mysurface<T>
{
public:
	T power;//just random number, default to 0.1 mm^-1

	__host__ __device__ powersurface(T power = 0.1, const vec3<T>& pos = vec3<T>(0, 0, 0), T diameter = 10) :
		mysurface(pos, diameter, 1), power(power)
	{
		LOG1("power surface created")
	}

	__host__ __device__ ~powersurface()
	{
		LOG1("power surface destructor called")
	}

	int size()
	{
		return sizeof(*this);
	}
};

template<typename T = float>
class quadricparam
{
public:
	T A, B, C, D, E, F, G, H, I, J; //implicit equation A*x^2+B*y^2+C*z^2+D*x*y+E*x*z+F*y*z+G*x+H*y+I*z+J = 0

	__host__ __device__ quadricparam(T A = 1, T B = 1, T C = 1, T D = 0, T E = 0, T F = 0, T G = 0, T H = 0, T I = 0, T J = 0) :
		A(A), B(B), C(C), D(D), E(E), F(F), G(G), H(H), I(I), J(J)
	{}
};

template<typename T = float>
class quadricsurface :public mysurface<T>
{
public:
	quadricparam<T> param;
	T n1, n2;

	__host__ __device__ quadricsurface(const quadricparam<T>& param = quadricparam<T>(1,1,1,0,0,0,0,0,0,-1),
		T n1 = 1, T n2 = 1, const vec3<T>& pos = vec3<T>(0, 0, 0), T diameter = 10):
		mysurface(pos, diameter, 1), param(param), n1(n1), n2(n2)
	{
		LOG1("quadric surface created")
	}

	__host__ __device__ ~quadricsurface()
	{
		LOG1("quadric surface destructor called")
	}

	//needs to overwrite this function in every sub class inorder for it to return proper result
	__host__ __device__ int size()
	{
		return sizeof(*this);
	}
};

//main tracing kernel
#ifdef nothing
template <typename T = float>
__global__ void tracer(raysegment<T>* inbundle, raysegment<T>* outbundle, const mysurface<T>* nextsurface)
{
	// get thread index
	int idx = threadIdx.x;
	
	//return if it is an inactive ray segment
	if (inbundle[idx].status == 0)
	{
		outbundle[idx] = inbundle[idx];
		return;
	}

	auto surfacetype = nextsurface->type;

    // coordinate transformation
	auto before = raysegment<MYFLOATTYPE>(inbundle[idx].pos - nextsurface->pos,inbundle[idx].dir);


	// intersection find 
	auto t = ((MYFLOATTYPE)0 - before.pos.z) / (before.dir.z);// in surface's own coordinate, the surface is at z = 0
	auto at = raysegment<MYFLOATTYPE>(before.pos + t * before.dir,before.dir);
	
	// determine if valid intersection
	if (norm(vec3<MYFLOATTYPE>(at.pos.x, at.pos.y, 0)) > (nextsurface->diameter) / 2)
	{
		inbundle[idx].status = 0;
		outbundle[idx] = inbundle[idx];
		return;
	}

	if (surfacetype == 1) // if next surface is a power surface
	{
		//surface transfer
		auto normalvec = vec3<MYFLOATTYPE>(0, 0, 1);
		auto radialvec = vec3<MYFLOATTYPE>(at.pos.x, at.pos.y, 0);
		auto binormal = normalize(cross(normalvec, radialvec));
		auto tangential = dot(at.dir, binormal)*binormal;
		auto radial = at.dir - tangential;
		auto u = acosf(dot(normalize(radial), normalize(-normalvec)));
		auto uprime = u - norm(radialvec)*((powersurface<MYFLOATTYPE>*)nextsurface)->power;

		auto newradial = norm(radial)*normalize(((-normalvec) + 
			normalize(radialvec)*((MYFLOATTYPE)tanf(uprime))));
		auto after = raysegment<MYFLOATTYPE>(at.pos, tangential + newradial);

		//printf("%d at u = %f, u' = %f\n", idx, u, uprime);

		// coordinate detransformation
		after.pos = after.pos + nextsurface->pos;

		// write results
		outbundle[idx] = after;
	}
	else if (surfacetype == 0) // if next surface is an image surface
	{
		// coordinate detransformation
		at.pos = at.pos + nextsurface->pos;
		at.status = 2;

		// write results
		outbundle[idx] = at;
	}

	

	/*printf("%d at t = %f at dir (%f,%f,%f), after dir (%f,%f,%f)\n", idx, t, at.dir.x, at.dir.y, 
		at.dir.z, after.dir.x, after.dir.y, after.dir.z );*/
}
#endif

//quadric tracer kernel, each block handles one bundle, each thread handles one ray
template<typename T = float>
__global__ void quadrictracer(raybundle<T>** d_inbundles, raybundle<T>** d_outbundles, int kernelparams[5])
{

	//TO DO: adapt this kernel to the new structure
	//get the indices
	int blockidx = blockIdx.x;
	int idx = threadIdx.x;

	//grab the correct in and out ray bundles
	raybundle<T>* inbundle = d_inbundles[blockidx];
	raybundle<T>* outbundle = d_outbundles[blockidx];

	//grab the correct ray of this thread
	raysegment<T> before = (inbundle->prays)[idx];

	//quit if ray is deactivated
	if (before.status == 0)
	{
		(outbundle->prays)[idx] = (inbundle->prays)[idx];
		return;
	}

	//TO DO: load the surface
	//test case
	auto pquad = new quadricsurface<MYFLOATTYPE>(quadricparam<MYFLOATTYPE>(1, 1, 1, 0, 0, 0, 0, 0, 0, -1));

	// copy to the shared memory
	__shared__ quadricsurface<MYFLOATTYPE> quadric;
	quadric = *pquad;

	//coordinate transformation
	before = quadric.coordinate_transform(before);

	/*
	__shared__ raysegment<MYFLOATTYPE> loadedbundle[bundlesize];
	loadedbundle[idx] = *pray;
	auto before = loadedbundle[idx];
	*/

	//find intersection

	//define references, else it will look too muddy
	MYFLOATTYPE &A = quadric.param.A,
		&B = quadric.param.B,
		&C = quadric.param.C,
		&D = quadric.param.D,
		&E = quadric.param.E,
		&F = quadric.param.F,
		&G = quadric.param.G,
		&H = quadric.param.H,
		&K = quadric.param.I, // in order not to mix with imaginary unit, due to the symbolic calculation in Maple
		&J = quadric.param.J;
	MYFLOATTYPE &p1 = before.pos.x,
		&p2 = before.pos.y,
		&p3 = before.pos.z,
		&d1 = before.dir.x,
		&d2 = before.dir.y,
		&d3 = before.dir.z;
	MYFLOATTYPE delta = - 4*A*B*d1*d1*p2*p2 + 8*A*B*d1*d2*p1*p2 - 4*A*B*d2*d2*p1*p1 
		- 4*A*C*d1*d1*p3*p3 + 8 * A*C*d1*d3*p1*p3 - 4 * A*C*d3*d3*p1*p1 - 4*A*F*d1*d1*p2*p3 
		+ 4*A*F*d1*d2*p1*p3 + 4*A*F*d1*d3*p1*p2 - 4*A*F*d2*d3*p1*p1 - 4 * B*C*d2*d2*p3*p3 
		+ 8*B*C*d2*d3*p2*p3 - 4*B*C*d3*d2*p2*p2 + 4*B*E*d1*d2*p2*p3 - 4*B*E*d1*d3*p2*p2 
		- 4*B*E*d2*d2*p1*p3 + 4*B*E*d2*d3*p1*p2 - 4*C*D*d1*d2*p3*p3 + 4*C*D*d1*d3*p2*p3 
		+ 4*C*D*d2*d3*p1*p3 - 4*C*D*d3*d3*p1*p2 + D*D*d1*d1*p2*p2 - 2*D*D*d1*d2*p1*p2 
		+ D*D*d2*d2*p1*p1 + 2*D*E*d1*d1*p2*p3 - 2*D*E*d1*d2*p1*p3 - 2*D*E*d1*d3*p1*p2 
		+ 2*D*E*d2*d3*p1*p1 - 2*D*F*d1*d2*p2*p3 + 2*D*F*d1*d3*p2*p2 + 2*D*F*d2*d2*p1*p3 
		- 2*D*F*d2*d3*p1*p2 + E*E*d1*d1*p3*p3 - 2*E*E*d1*d3*p1*p3 + E*E*d3*d3*p1*p1 
		+ 2*E*F*d1*d2*p3*p3 - 2*E*F*d1*d3*p2*p3 - 2*E*F*d2*d3*p1*p3 + 2*E*F*d3*d3*p1*p2 
		+ F*F*d2*d2*p3*p3 - 2*F*F*d2*d3*p2*p3 + F*F*d3*d3*p2*p2 - 4*A*H*d1*d1*p2 
		+ 4*A*H*d1*d2*p1 - 4*A*K*d1*d1*p3 + 4*A*K*d1*d3*p1 + 4*B*G*d1*d2*p2 - 4*B*G*d2*d2*p1 
		- 4*B*K*d2*d2*p3 + 4*B*K*d2*d3*p2 + 4*C*G*d1*d3*p3 - 4*C*G*d3*d3*p1 + 4*C*H*d2*d3*p3 
		- 4*C*H*d3*d3*p2 + 2*D*G*d1*d1*p2 - 2*D*G*d1*d2*p1 - 2*D*H*d1*d2*p2 
		+ 2*D*H*d2*d2*p1 - 4*D*K*d1*d2*p3 + 2*D*K*d1*d3*p2 + 2*D*K*d2*d3*p1 + 2*E*G*d1*d1*p3 
		- 2*E*G*d1*d3*p1 + 2*E*H*d1*d2*p3 - 4*E*H*d1*d3*p2 + 2*E*H*d2*d3*p1 - 2*E*K*d1*d3*p3 
		+ 2*E*K*d3*d3*p1 + 2*F*G*d1*d2*p3 + 2*F*G*d1*d3*p2 - 4*F*G*d2*d3*p1 + 2*F*H*d2*d2*p3 
		- 2*F*H*d2*d3*p2 - 2*F*K*d2*d3*p3 + 2*F*K*d3*d3*p2 - 4*A*J*d1*d1 - 4*B*J*d2*d2 
		- 4*C*J*d3*d3 - 4*D*J*d1*d2 - 4*E*J*d1*d3 - 4*F*J*d2*d3 + G*G*d1*d1 + 2*G*H*d1*d2 
		+ 2*G*K*d1*d3 + H*H*d2*d2+ 2*H*K*d2*d3 + K*K*d3*d3;
	MYFLOATTYPE deno = -2 * (A*d1*d1 + B*d2*d2 + C*d3*d3 + D*d1*d2 + E*d1*d3 + F*d2*d3);
	MYFLOATTYPE beforedelta = 2 * A*d1*p1 + 2 * B*d2*p2 + 2 * C*d3*p3 + D * (d1*p2 + d2 * p1) + E * (d1*p3 + d3 * p1) + F * (d2*p3 + d3 * p2) + G * d1 + H * d2 + K * d3;
	MYFLOATTYPE t, t1, t2;
	t1 = (delta >= 0) ? (beforedelta + sqrtf(delta)) / deno : INFINITY;
	t2 = (delta >= 0) ? (beforedelta - sqrtf(delta)) / deno : INFINITY;

	//pick the nearest positive intersection
	if (t1 >= 0 && t2 >= 0)
		t = (t1 < t2) ? t1 : t2;
	else if (t1 < 0 && t2 >= 0)
		t = t2;
	else if (t2 < 0 && t1 >= 0)
		t = t1;
	else
		t = INFINITY;

	// if there is an intersection
	if (t < INFINITY)
	{
		auto at = raysegment<MYFLOATTYPE>(before.pos + t * before.dir, before.dir);

		//is the intersection within hit box ? if not, then deactivate the ray
		if (!quadric.hitbox(at.pos)) goto deactivate_ray;
		
		// if it is a refractive surface, do refractive ray transfer
		if (quadric.type == 1)
		{
			//refractive surface transfer
			auto after = raysegment<MYFLOATTYPE>(at.pos, at.dir);
			MYFLOATTYPE &x = at.pos.x,
				&y = at.pos.y,
				&z = at.pos.z;
			auto surfnormal = normalize(vec3<MYFLOATTYPE>(2 * A*x + D * y + E * z + G, 2 * B*y + D * x + F * z + H, 2 * C*z + E * x + F * y + K));

			auto ddotn = dot(at.dir, surfnormal);
			ddotn = (ddotn < 0) ? ddotn : -ddotn; // so that the surface normal and ray are in opposite direction

			MYFLOATTYPE factor1 = 1 - quadric.n1*quadric.n1 / (quadric.n2*quadric.n2)
				*(1 - ddotn * ddotn);
			if (factor1 < 0)
			{
#ifdef _DEBUGMODE2
				printf("something is wrong with transfer refractive vectors");
#endif
				goto deactivate_ray;
			}

			after.dir = quadric.n1*(at.dir - surfnormal * ddotn) / quadric.n2 - surfnormal * (MYFLOATTYPE)sqrtf(factor1);

			//coordinate detransformation, write out result
			after = quadric.coordinate_detransform(after);
			(outbundle->prays)[idx] = after;
		}
		// else if it is an image surface
		else if (quadric.type == 0)
		{
			//coordinate detranformation of at and write out result
			at = quadric.coordinate_detransform(at);
			at.status = 2;
			(outbundle->prays)[idx] = at;
		}
	}
	//else there is no intersection, deactivate the ray
	else
		goto deactivate_ray;

deactivate_ray:
	{
		// TO DO: write out ray status, copy input to output
		(outbundle->prays)[idx] = (inbundle->prays)[idx];
		(outbundle->prays)[idx].status = 0;
	};


	/*
	printf("delta = %f ,beforedelta = %f ,deno = %f \n", delta, beforedelta, deno);
	printf("t1 = %f ,t2 = %f ,t = %f\n", t1, t2,t);
	printf("%d at t = %f ,pos = (%f,%f,%f), surfnormal (%f,%f,%f), factor1 = %f, at dir (%f,%f,%f), after dir (%f,%f,%f)\n", 
		idx, t, at.pos.x, at.pos.y, at.pos.z, 
		surfnormal.x, surfnormal.y,surfnormal.z,factor1,
		at.dir.x, at.dir.y, at.dir.z,
		after.dir.x, after.dir.y, after.dir.z );
	*/


	//clean up the test case
	delete pquad;
}

#ifdef nothing
class test
{
public:
	int t;
	void method()
	{
		LOG1("test method")
	}
	test(int t = 0):t(t)
	{
		LOG1("test object created")
	}
	~test()
	{
		LOG1("test destructor called")
	}
};
#endif

int main()
{
	LOG1("this is main program");



	//testing bundle creation
#ifdef something
	//creating an array of ray bundles
	float diam = 10;
	int numofsurfaces = 3;
	raybundle<MYFLOATTYPE>* bundles = new raybundle<MYFLOATTYPE>[numofsurfaces + 1];
	for (int i = 0; i < numofsurfaces + 1; i++)
	{
		bundles[i] = raybundle<MYFLOATTYPE>();
	}

	//initialize the first bundle
	bundles[0].init_2D_dualpolar(vec3<MYFLOATTYPE>(0, 0, 20), -3, 3, -2, 2, 0.707);
#endif



#ifdef nothing
	//create event for timing
	hipEvent_t start, stop;
	CUDARUN(hipEventCreate(&start));
	CUDARUN(hipEventCreate(&stop));

	//start timing 
	CUDARUN(hipEventRecord(start, 0));

	//launch kernel
	{tester << <1, 1 >> > ();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error at file %s line %d, ", __FILE__, __LINE__);
		fprintf(stderr, "code %d, reason %s\n", cudaStatus, hipGetErrorString(cudaStatus));
	}
	}

	//kernel finished, stop timing, print out elapsed time
	CUDARUN(hipEventRecord(stop, 0));
	CUDARUN(hipEventSynchronize(stop));
	float elapsedtime;
	CUDARUN(hipEventElapsedTime(&elapsedtime, start, stop));
	LOG2("kernel run time: " << elapsedtime << " ms\n");
#endif

#ifdef nothing
	//create event for timing
	hipEvent_t start, stop;
	CUDARUN(hipEventCreate(&start));
	CUDARUN(hipEventCreate(&stop));

	//set up the surfaces manually !!!!
	float diam = 10;
	int numofsurfaces = 3;
	mysurface<MYFLOATTYPE>** psurfaces = new mysurface<MYFLOATTYPE>*[numofsurfaces];
	psurfaces[0] = new powersurface<MYFLOATTYPE>(-0.1, vec3<MYFLOATTYPE>(0, 0, 13),diam);
	psurfaces[1] = new powersurface<MYFLOATTYPE>(0.2, vec3<MYFLOATTYPE>(0, 0, 11), diam);
	psurfaces[2] = new mysurface<MYFLOATTYPE>(vec3<MYFLOATTYPE>(0, 0, 0), diam);
	

	//create ray bundles for tracing
	raysegment<MYFLOATTYPE>** bundles = new raysegment<MYFLOATTYPE>*[numofsurfaces+1];
	for (int i = 0; i < numofsurfaces+1; i++)
	{
		bundles[i] = new raysegment<MYFLOATTYPE>[bundlesize];
	}


	//set up the original bundle, manually
	for (int i = 0; i < bundlesize; i++)
	{
		static float step = diam / 32;
		static float start = -(diam / 2) + (step / 2);
		bundles[0][i] = raysegment<MYFLOATTYPE>(vec3<MYFLOATTYPE>(start + step * i, 0, 20), vec3<MYFLOATTYPE>(0, 0, -1));
		//LOG2(i <<" "<< bundle[i]);
	}


	// allocate device memory for 2 bundles
	size_t batchsize = bundlesize * sizeof(raysegment<MYFLOATTYPE>);

	raysegment<MYFLOATTYPE>* d_inbundle;
	CUDARUN(hipMalloc((void**)&d_inbundle, batchsize));
	raysegment<MYFLOATTYPE>* d_outbundle;
	CUDARUN(hipMalloc((void**)&d_outbundle, batchsize));


	//allocate device memory for surfaces
	void** d_psurfaces = new void*[numofsurfaces];

	for (int i = 0; i < numofsurfaces; i++)
	{
		CUDARUN(hipMalloc((void**)&(d_psurfaces[i]), psurfaces[i]->size()));
	}


	//start timing 
	CUDARUN(hipEventRecord(start, 0));


	//copy original bundle data to device
	CUDARUN(hipMemcpy(d_inbundle, bundles[0], batchsize, hipMemcpyHostToDevice));


	//copy surfaces data to device
	for (int i = 0; i < numofsurfaces; i++)
	{
		CUDARUN(hipMemcpy(d_psurfaces[i], psurfaces[i], psurfaces[i]->size(), hipMemcpyHostToDevice));
	}
	
	// launch kernel, copy result out, swap memory

	for (int i = 0; i < numofsurfaces; i++)
	{
		tracer <<<1, 32 >>> (d_inbundle, d_outbundle, static_cast<mysurface<MYFLOATTYPE>*>(d_psurfaces[i]));
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Error at file %s line %d, ", __FILE__, __LINE__);
			fprintf(stderr, "code %d, reason %s\n", cudaStatus, hipGetErrorString(cudaStatus));
		}
		CUDARUN(hipMemcpy(bundles[i+1], d_outbundle, batchsize, hipMemcpyDeviceToHost));
		swap(d_inbundle, d_outbundle);
	}


	//kernel finished, stop timing, print out elapsed time
	CUDARUN(hipEventRecord(stop, 0));
	CUDARUN(hipEventSynchronize(stop));
	float elapsedtime;
	CUDARUN(hipEventElapsedTime(&elapsedtime, start, stop));
	LOG2("kernel run time: " << elapsedtime << " ms\n");

	//writing results out
	for (int i = 0; i < bundlesize; i++)
	{
		LOG2("ray " << i);
		for (int j = 0; j < numofsurfaces+1; j++)
		{
			switch (bundles[j][i].status)
			{
			case 0:
				LOG2(" deactivated")
				break;
			case 1:
				LOG2(" " << bundles[j][i])
				break;
			case 2:
				if (bundles[j-1][i].status != 0)
					LOG2(" " << bundles[j][i] << " done")
				break;
			}
		}
		LOG2("\n");
	}

	//destroy cuda timing events
	CUDARUN(hipEventDestroy(start));
	CUDARUN(hipEventDestroy(stop));

	// free device heap momory
	hipFree(d_inbundle);
	hipFree(d_outbundle);
	for (int i = 0; i < numofsurfaces; i++)
	{
		hipFree(d_psurfaces[i]);
	}
	delete[] d_psurfaces;

	//free host heap momory
	for (int i = 0; i < numofsurfaces; i++)
	{
		delete psurfaces[i];
	}
	delete[] psurfaces;
	
	for (int i = 0; i < numofsurfaces+1; i++)
	{
		delete[] bundles[i];
	}

	delete[] bundles;
#endif
}
